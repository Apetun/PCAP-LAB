#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>  // For sinf

__global__ void computeSine(float *input, float *output, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index

    if (idx < N) {
        output[idx] = sinf(input[idx]);  // Compute sine of the angle
    }
}

int main(void) {
    int N = 1000;  // Size of the array
    int size = N * sizeof(float);

    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(size);
    float *d_input, *d_output;

    // Initialize host input array with some angles in radians
    for (int i = 0; i < N; i++) {
        h_input[i] = (float)(i * M_PI / 180.0);  // Angles in radians (1 degree step)
    }

    // Allocate device memory
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Define block size and number of blocks
    int THREADS_PER_BLOCK = 256;
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    computeSine<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, N);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Print the first 10 results for verification
    printf("First 10 Sine Values:\n");
    for (int i = 0; i < (N < 10 ? N : 10); i++) {
        printf("sin(%f) = %f\n", h_input[i], h_output[i]);
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}
